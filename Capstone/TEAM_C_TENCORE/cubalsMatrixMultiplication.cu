
/*
병렬프로그래밍 c_team
 10월 29일 과제
 cuda를 이용하여 행렬 A*B+C 연산 구현
 +cublas 라이브러리를 사용할 것
*/

#include <stdio.h> 
#include <stdlib.h> 
#include <hip/hip_runtime.h> 
#include "hipblas.h"

#define IDX2C(i,j,ld) (((j)*(ld))+( i ))  // 메모리 index
#define m 6 // a 는 mxk 행렬 
#define n 4 // b 는 kxn 행렬 
#define k 5 // c 는 mxn 행렬

int main(void) {
 hipError_t cudaStat; // cudaMalloc status 
 hipblasStatus_t stat; // CUBLAS functions status 
 hipblasHandle_t handle; // CUBLAS context
 
 int i, j; // i는 행의 index j는 열의 index
 
 // 행열 메모리 주소를 가리킬 포인터
 float* a;
 float* b;
 float* c;
 
 // 메모리를 행렬의 크기만큼 할당한다.
 a = (float*)malloc(m*k * sizeof(float)); // host memory for a
 b = (float*)malloc(k*n * sizeof(float)); // host memory for b 
 c = (float*)malloc(m*n * sizeof(float)); // host memory for c

 // cublas 행렬곱 연산은 cloumn major이기 때문에
 // 행렬 a를 열 기준으로 값을 저장한다.
 int ind = 11; // 행렬의 1행 1열의 값
 for (j = 0; j < k; j++) {
  for (i = 0; i < m; i++) {
   a[IDX2C(i, j, m)] = (float)ind++;
  }
 }
 // 11,17,23,29,35 
 // 12,18,24,30,36 
 // 13,19,25,31,37 
 // 14,20,26,32,38
 // a 행렬 출력
 printf("a:\n");
 for (i = 0; i < m; i++) {
  for (j = 0; j < k; j++)
  {
   printf("%5.0f", a[IDX2C(i, j, m)]);
  }
  printf("\n");
 }
 // b행렬 
 ind = 11; // 행렬 1행 1열의 값 
 for (j = 0; j < n; j++) {
  for (i = 0; i < k; i++) {
   b[IDX2C(i, j, k)] = (float)ind++;
  }
 }
 //b:
 //11,16,21,26
 //12,17,22,27
 //13,18,23,28
 //14,19,24,29
 //15,20,25,30

 printf("b:\n"); 
 for (i = 0; i < k; i++) { 
  for (j = 0; j < n; j++) {
   printf("%5.0f", b[IDX2C(i, j, k)]); 
  } 
  printf("\n");
 }
 
 ind = 11; // 1행 1열의 값
 for(j=0;j<n;j++){
   for(i=0;i<m;i++){
    c[IDX2C(i,j,m)]=(float)ind++;
  } 
 }
 
 //c:
 //11,17,23,29
 //12,18,24,30
 //13,19,25,31
 //14,20,26,32
 //15,21,27,33
 //16,22,28,34
 
 printf("c:\n"); 
 for (i = 0; i < m; i++) { 
  for (j = 0; j < n; j++) { 
   printf("%5.0f", c[IDX2C(i, j, m)]); 
  } 
  printf("\n"); 
 }
 
 // gpu로 값을 넘주는 작업
 float* d_a; 
 float* d_b; 
 float* d_c;
 
 // 행렬 크기만큼 gpu 메모리 할당  
 cudaStat=hipMalloc((void**)&d_a,m*k*sizeof(*a)); 
 cudaStat=hipMalloc((void**)&d_b,k*n*sizeof(*b)); 
 cudaStat=hipMalloc((void**)&d_c,m*n*sizeof(*c));

 stat = hipblasCreate(&handle); // cublas 초기화
 
 // 행렬의 값을 gpu메모리로 복사
 stat = hipblasSetMatrix(m, k, sizeof(*a), a, m, d_a, m);//a -> d_a 
 stat = hipblasSetMatrix(k,n,sizeof(*b),b,k,d_b,k);//b -> d_b 
 stat = hipblasSetMatrix(m,n,sizeof(*c),c,m,d_c,m);//c -> d_c
 // matrix -matrix multiplication: d_c = al*d_a*d_b + bet*d_c 
 // d_a -mxk matrix , d_b -kxn matrix , d_c -mxn matrix;
 
 // 사용하는 함수는 C=al*A*B+C 결과를 가진다 
 // 따라서 C가 0행렬이라면 A*B의 결과만
 // C의 값이 존재한다면 A*B+C의 결과르 가진다. 
 
 // al,bet는 행렬의 scalar 이다.
 float al = 1.0f; // al=1
 float bet=1.0f; //bet=1
 
 stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &al, d_a, m, d_b, k, &bet, d_c, m);
 stat = hipblasGetMatrix(m, n, sizeof(*c), d_c, m, c, m); //cp d_c->c 
 
 printf("c after Sgemm :\n"); 
 for(i=0;i<m;i++){ 
  for(j=0;j<n;j++){ 
   printf("%7.0f",c[IDX2C(i,j,m)]); //print c after Sgemm 
  } 
  printf("\n"); 
 } 
 
 // gpu 메모리 헤재
 hipFree(d_a); 
 hipFree(d_b); 
 hipFree(d_c); 
 
 hipblasDestroy(handle); //쿠다 명령어 삭제
 
 // cpu 메모리 헤제 
 free(a); 
 free(b); 
 free(c); 
 return 0;
}
