
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>



 // Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}
 #define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}
 #define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}
 #include <mma.h>
using namespace nvcuda;
 // Must be multiples of 16 for wmma code to work
#define MATRIX_M 16384
#define MATRIX_N 16384
#define MATRIX_K 16384



 __global__ void convertFp32ToFp16 (half *out, float *in, int n) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      out[idx] = in[idx];
   }
}


 int main(int argc, char* argv[]) {
   float *a_fp32;
   float *b_fp32;
   half *a_fp16;
   half *b_fp16;
    float *c;
   float *c_cublas;
   float *c_cuda;
    float *c_host_cublas;
   float *c_host_cuda;

   
   hiprandGenerator_t gen;
   hipblasHandle_t cublasHandle;
   
   hipEvent_t startCUDA;
   hipEvent_t stopCUDA;
   
   hipEvent_t startcublas;
   hipEvent_t stopcublas;
   
   cudaErrCheck(hipEventCreate(&startCUDA));
   cudaErrCheck(hipEventCreate(&stopCUDA));
   
   cudaErrCheck(hipEventCreate(&startcublas));
   cudaErrCheck(hipEventCreate(&stopcublas));
   
   
   cublasErrCheck(hipblasCreate(&cublasHandle));
   
   
   // 쿠다 코어로 변경 1번 CUBLAS_TENSOR_OP_MATH를 CUBLAS_DEFAULT_MATH로 변경해준다.
   cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH));// 
   
// 메모리 할당
   cudaErrCheck(hipMalloc((void**)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
   cudaErrCheck(hipMalloc((void**)&b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));

    cudaErrCheck(hipMalloc((void**)&c, MATRIX_M * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&c_cublas, MATRIX_M * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&c_cuda, MATRIX_M * MATRIX_N * sizeof(float)));

    c_host_cublas = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
   c_host_cuda = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));

    curandErrCheck(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
   curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL));

    curandErrCheck(hiprandGenerateUniform(gen, a_fp32, MATRIX_M * MATRIX_K));
   curandErrCheck(hiprandGenerateUniform(gen, b_fp32, MATRIX_K * MATRIX_N));

    // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
   convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (a_fp16, a_fp32, MATRIX_M * MATRIX_K);
   convertFp32ToFp16 <<< (MATRIX_K * MATRIX_N + 255) / 256, 256 >>> (b_fp16, b_fp32, MATRIX_K * MATRIX_N);
    curandErrCheck(hiprandGenerateUniform(gen, c, MATRIX_M * MATRIX_N));
   
   curandErrCheck(hiprandDestroyGenerator(gen));
   
   cudaErrCheck(hipMemcpy(c_cublas, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));
   cudaErrCheck(hipMemcpy(c_cuda, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));
    float alpha = 2.0f;
   float beta = 2.0f;
    printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
   
   
   
   // Now using cuBLAS with CUDA
   printf("Running with cuBLAS with Cuda Core\n");
   cudaErrCheck(hipEventRecord(startCUDA));
   // 쿠다 코어로 변경 2번
   // 쿠다 코어 이용시에 CUUBLAS_GEMM_DFALT_TENSOR_OP을 CUBLAS_GEMM_DEFALT 로 변경해준다. 
   cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                MATRIX_M, MATRIX_N, MATRIX_K, 
                &alpha,
                a_fp16, HIP_R_16F, MATRIX_M,
                b_fp16, HIP_R_16F, MATRIX_K,
                &beta, 
                c_cuda, HIP_R_32F, MATRIX_M,
                HIP_R_32F, HIPBLAS_GEMM_DEFAULT));
   cudaErrCheck(hipEventRecord(stopCUDA));
    // Error checking
  

   cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));// 

   // Now using cuBLAS with Tensor
   printf("Running with cuBLAS with Tensor Core\n");
   cudaErrCheck(hipEventRecord(startcublas));
   // 쿠다 코어로 변경 2번
   // 쿠다 코어 이용시에 CUUBLAS_GEMM_DFALT_TENSOR_OP을 CUBLAS_GEMM_DFALT 로 변경해준다. 
   cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                MATRIX_M, MATRIX_N, MATRIX_K, 
                &alpha,
                a_fp16, HIP_R_16F, MATRIX_M,
                b_fp16, HIP_R_16F, MATRIX_K,
                &beta, 
                c_cublas, HIP_R_32F, MATRIX_M,
                HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
   cudaErrCheck(hipEventRecord(stopcublas));
    // Error checking
   printf("\nChecking results...\n");

   cudaErrCheck(hipMemcpy(c_host_cuda, c_cuda, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
   cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
   
   
  
  
      printf("Results.\n\n");
      float cudaTime;
      float cublasTime;
      cudaErrCheck(hipEventSynchronize(stopCUDA));
      cudaErrCheck(hipEventSynchronize(stopcublas));
      cudaErrCheck(hipEventElapsedTime(&cudaTime, startCUDA, stopCUDA));
      cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
      
     // TFLOPS 계산 결과 출력
      printf("cuda took %fms\n", cudaTime);
      printf("[+] TFLOPS: %.2f\n", ((double)MATRIX_M * MATRIX_N * MATRIX_K * 2) / cudaTime / 1e9);
      printf("tensor took %fms\n", cublasTime);
      printf("[+] TFLOPS: %.2f\n", ((double)MATRIX_M * MATRIX_N * MATRIX_K * 2) / cublasTime / 1e9);
   
       printf("\nCUBALS WITH CUDA OR TENSOR CORE CODE !\n\n");
   
   
      
   
   cudaErrCheck(hipEventDestroy(startCUDA));
   cudaErrCheck(hipEventDestroy(stopCUDA));
    cudaErrCheck(hipEventDestroy(startcublas));             
   cudaErrCheck(hipEventDestroy(stopcublas));
   
   cudaErrCheck(hipFree(a_fp32));
   cudaErrCheck(hipFree(b_fp32));
   cudaErrCheck(hipFree(a_fp16));
   cudaErrCheck(hipFree(b_fp16));
    cudaErrCheck(hipFree(c));
   cudaErrCheck(hipFree(c_cublas));
   cudaErrCheck(hipFree(c_cuda));
   
   free(c_host_cublas);
   free(c_host_cuda);
    cudaErrCheck(hipDeviceReset());
   return 0;
}
