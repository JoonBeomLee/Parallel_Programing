#include "hip/hip_runtime.h"
/*
https://www.cise.ufl.edu/~sahni/papers/strassen.pdf
슈트라센 알고리즘을 쿠다로 구현한 코드입니다. 
*/

__device__ void update2(float *a, float b, float *c)
{
   for (int i = 0; i < 16; i++)
      c[i] += a[i * 4] * b;
}

__global__ void GPU8 (float *a, float *b, float *c, int n)
{// thread code to compute one column of a 16 x 128 sub-matrix of c
 // use shared memory to hold the transpose of a
 // 16 x 64 sub-matrix of 1 x 4 sub-vectors of a
    __shared__ float as[16][65];
    // registers for column of c sub-matrix
    float cr[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    int nDiv64 = n/64;
    int sRow = threadIdx.y;
    int sRow4 = sRow*4;
    int sCol = threadIdx.x;
    int tid = sRow*16+sCol.x;
    int aNext = (16*blockIdx.y+sRow)*n+sCol*4;
    int bNext = 128*blockIdx.x + tid;
    int cNext = 16*blockIdx.y*n + 128*blockIdx.x + tid;
    int nTimes2 = 2*n;
    int nTimes3 = 3*n;
    int nTimes4 = 4*n;
    a += aNext;
    b += bNext;
    c += cNext;
    float4 *a4 = (float4 *)a;
    for (int i = 0; i < nDiv64; i++)
    {
        *( (float4 *)(&as[sCol][sRow4]) ) = a4[0];
        *( (float4 *)(&as[sCol][sRow4+32]) ) = a4[nTimes2];
        __syncthreads(); // wait for read to complete
        
        float br0 = b[0];
        float br1 = b[n];
        float br2 = b[nTimes2];
        float br3 = b[nTimes3];
      b += nTimes4;
        #pragma unroll
        for (int k = 0; k < 15; k++)
        {
            update2 (&as[k][0], br0, cr); br0 = b[0];
            update2 (&as[k][1], br1, cr); br1 = b[n];
            update2 (&as[k][2], br2, cr); br2 = b[nTimes2];
            update2 (&as[k][3], br3, cr); br3 = b[nTimes3];
            b+= nTimes4;
        }
        update2 (&as[15][0], br0, cr);
        update2 (&as[15][1], br1, cr);
        update2 (&as[15][2], br2, cr);
        update2 (&as[15][3], br3, cr);
        a4 += 16;
        __syncthreads(); // wait for computation to complete
    }
    for (int j = 0; j < 16; j++)
    {
      c[0] = cr[j];
       c += n; }
}


__global__ void add (float *d_A, float *d_B, float *d_C, int widthA, int widthB, int widthC)
{
    int startA = blockIdx.x*64 + threadIdx.x*2 + (blockIdx.y*8 + threadIdx.y)*widthA;
    int startB = blockIdx.x*64 + threadIdx.x*2 + (blockIdx.y*8 + threadIdx.y)*widthB;
    int startC = blockIdx.x*64 + threadIdx.x*2 + (blockIdx.y*8 + threadIdx.y)*widthC;
    float2 tempA = *(float2 *)(d_A+startA);
    float2 tempB = *(float2 *)(d_B+startB);
    tempA.x += tempB.x;
    tempA.y += tempB.y;
    *(float2 *)(d_C+startC) = tempA;
}
