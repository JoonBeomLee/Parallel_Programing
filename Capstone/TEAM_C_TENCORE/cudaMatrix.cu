
#include <stdio.h> 
#include <stdlib.h> 
#include <hip/hip_runtime.h> 
#include <iostream>

#define IDX2C(rowlength,i,j) ((rowlength*i)+j) // 행렬의 index 구하는 방식 

#define m 2
#define k 3
#define n 2

using namespace std;


// 쓰레드의 인덱스 만큼 행렬의 덧샘을 실행하는 함수 
__global__ void add(int *c , int *d){
    int tid=threadIdx.x;
    d[tid]+=c[tid];
}


// 쓰레드의 인덱스와 블록 인덱스를 사용하여 
// 행렬의 곱샘 연산으 병렬화 합니다.
__global__ void multi(int *a , int *b,int *d){
    int tid=threadIdx.x; 
    int bid=blockIdx.x;
    
    for(int l=0;l<k; l++){
        d[IDX2C(n,tid,bid)]+=a[IDX2C(k,tid,l)]*b[IDX2C(n,l,bid)];
    }     
}


int main(){
    int *a;
    int *b;
    int *c;
    int *d;
    int i,j;

    //device 메모리
    int *d_a,*d_b,*d_c,*d_d;
    
    // 행렬 메모리 할당 
    a=(int*)malloc( m*k * sizeof(int) ); 
    b=(int*)malloc( k*n * sizeof(int) ); 
    c=(int*)malloc( m*n * sizeof(int) ); 
    d=(int*)malloc( m*n * sizeof(int) ); 
    

    


    int value_a[m*k]={1,0,-3,-2,4,1};
    int value_b[k*n]={2,-1,3,0,-5,2};
    int value_c[m*n]={3,-1,-2,2};


    // a,b,c행렬의 값을 넣고 확인한다. 
    cout<<"a:\n";
    for(i=0; i<m*k; i++){
        a[i]=value_a[i];
    }

    for(i=0; i<m; i++){
        for(j=0;j<k;j++){
            cout<<(a[ IDX2C(k,i,j) ])<<" ";
        }
        cout<<endl;
    }

    cout<<"b:\n";
    for(i=0; i<k*n; i++){
        b[i]=value_b[i];
    }

    for(i=0; i<k; i++){
        for(j=0;j<n;j++){
            cout<<(b[ IDX2C(n,i,j) ])<<" ";
        }
        cout<<endl;
    }

    cout<<"c:\n";
    for(i=0; i<m*n; i++){
        c[i]=value_c[i];
    }

    for(i=0; i<m; i++){
        for(j=0;j<n;j++){
            cout<<(c[ IDX2C(n,i,j) ])<<" ";
        }
        cout<<endl;
    }

    // 결과는 0으로 초기화
    for(i=0; i<m*n; i++){
        d[i]=0;
    }

    // cuda 메모리 할당
    hipMalloc( (void**)&d_a , m*k*sizeof(int) ) ;
    hipMalloc( (void**)&d_b , k*n*sizeof(int) ) ;
    hipMalloc( (void**)&d_c , m*n*sizeof(int) ) ;
    hipMalloc( (void**)&d_d , m*n*sizeof(int) ) ;

    // device로 행렬값 전달 
    hipMemcpy( d_a,a,m*k*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpy( d_b,b,k*n*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpy( d_c,c,m*n*sizeof(int),hipMemcpyHostToDevice );
    


    //열의 수 만큼 블록을 행의 수만큼 쓰레드를 생성한다.
    multi<<<k,m>>>(d_a,d_b,d_d);

    //메모리 가지고 오기
    hipMemcpy( d,d_d,m*n*sizeof(int),hipMemcpyDeviceToHost );

    cout<<"A*B 결과 d:\n";


    for(i=0; i<m; i++){
        for(j=0;j<n;j++){
            cout<<(d[ IDX2C(n,i,j) ])<<" ";
        }
        cout<<endl;
    }


    //행렬의 원소 수 만큼 쓰레드를 생성 후 덧샘
    
    add<<<1,m*n>>>(d_c,d_d);
    
    //메모리 가지고 오기
    hipMemcpy( d,d_d,m*n*sizeof(int),hipMemcpyDeviceToHost );

    cout<<"A*B+C 결과 d:\n";


    for(i=0; i<m; i++){
        for(j=0;j<n;j++){
            cout<<(d[ IDX2C(n,i,j) ])<<" ";
        }
        cout<<endl;
    }
    
    //device 메모리 헤제
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);



    // host 메모리 헤제

    free(a);
    free(b);
    free(c);
    free(d);

    return 0;
}
